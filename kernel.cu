#include "hip/hip_runtime.h"
#include "kernel.h"
#include <stdio.h>
#define TPB 32

// Set of functions to be performed on the device
__device__ float setValue(float val)
{
  return val;
}

__device__ float add(float num1, float num2)
{
  return num1 + num2;
}

__device__ float mult(float num1, float num2)
{
  return num1 * num2;
}


// Set of kernel functions
__global__ void setValKernel(float *d_out, float value)
{
  const int i = blockIdx.x*blockDim.x + threadIdx.x;
  d_out[i] = setValue(value);
}

__global__ void scalarMultKernel(float *d_out, float scalar, float *d_in)
{
  const int i = blockIdx.x*blockDim.x + threadIdx.x;
  d_out[i] = mult(scalar, d_in[i]);
}

__global__ void componentAddKernel(float *d_out, float *d_in, float *array2)
{
  const int i = blockIdx.x*blockDim.x + threadIdx.x;
  d_out[i] = add(d_in[i],array2[i]);
}

__global__ void linearFunctionKernel(float *d_out, float scalar, float *d_in, float *array2)
{
  const int i = blockIdx.x*blockDim.x + threadIdx.x;
  d_out[i] = add(mult(scalar, d_in[i]), array2[i]);
}

__global__ void componentMultKernel(float *d_out, float *d_in, float *array2)
{
  const int i = blockIdx.x*blockDim.x + threadIdx.x;
  d_out[i] = mult(d_in[i],array2[i]);
}

__global__ void sumPartsKernel(float *accum, float *d_in)
{
  const int i = blockIdx.x*blockDim.x + threadIdx.x;
  *accum += d_in[i];
}


// Creates arrays that hold answers and copy to host
//WORKS AS INTENDED
void setValArray(float *out, float val, int len)
{
  // Pointers to device arrays
  float *d_out = 0;

  // Allocate memory for device array
  hipMalloc(&d_out, len*sizeof(float));

  // Launch kernel to compute and store
  setValKernel<<<len/TPB, TPB>>>(d_out, val);
  
  // Copy from device to host
  hipMemcpy(out, d_out, len*sizeof(float), hipMemcpyDeviceToHost);

  // Free the memory
  hipFree(d_out);
}


//WORKS AS INTENDED
void compAddArray(float *out, float *in, float *array2, int len)
{
  // Pointers to device arrays
  float *d_in = 0;
  float *d_out = 0;
  float *d_array2 = 0;

  // Allocate memory for device arrays
  hipMalloc(&d_in, len*sizeof(float));
  hipMalloc(&d_out, len*sizeof(float));
  hipMalloc(&d_array2, len*sizeof(float));

  // Copy from host to device
  hipMemcpy(d_in, in, len*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_array2, array2, len*sizeof(float), hipMemcpyHostToDevice);

  // Launch kernel to compute and store
  componentAddKernel<<<len/TPB, TPB>>>(d_out, d_in, d_array2);
  
  // Copy from device to host
  hipMemcpy(out, d_out, len*sizeof(float), hipMemcpyDeviceToHost);

  // Free the memory
  hipFree(d_in);
  hipFree(d_out);
  hipFree(d_array2);
}

//WORKS AS INTENDED
void linearArray(float *out, float *in_1, float *in_2, float scalar, int len)
{
  // Pointers to device arrays
  float *d_in_1 = 0;
  float *d_in_2 = 0;
  float *d_out = 0;

  // Allocate memory for device arrays
  hipMalloc(&d_in_1, len*sizeof(float));
  hipMalloc(&d_in_2, len*sizeof(float));
  hipMalloc(&d_out, len*sizeof(float));

  // Copy from host to device
  hipMemcpy(d_in_1, in_1, len*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_in_2, in_2, len*sizeof(float), hipMemcpyHostToDevice);

  // Launch kernel to compute and store
  linearFunctionKernel<<<len/TPB, TPB>>>(d_out, scalar, d_in_1, d_in_2);
  
  // Copy from device to host
  hipMemcpy(out, d_out, len*sizeof(float), hipMemcpyDeviceToHost);

  // Free the memory
  hipFree(d_in_1);
  hipFree(d_in_2);
  hipFree(d_out);
}

//WORKS AS INTENDED
void compMultArray(float *out, float *in_1, float *in_2, int len)
{
  // Pointers to device arrays
  float *d_in_1 = 0;
  float *d_in_2 = 0;
  float *d_out = 0;

  // Allocate memory for device arrays
  hipMalloc(&d_in_1, len*sizeof(float));
  hipMalloc(&d_in_2, len*sizeof(float));
  hipMalloc(&d_out, len*sizeof(float));

  // Copy from host to device
  hipMemcpy(d_in_1, in_1, len*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_in_2, in_2, len*sizeof(float), hipMemcpyHostToDevice);

  // Launch kernel to compute and store
  componentMultKernel<<<len/TPB, TPB>>>(d_out, d_in_1, d_in_2);
  
  // Copy from device to host
  hipMemcpy(out, d_out, len*sizeof(float), hipMemcpyDeviceToHost);

  // Free the memory
  hipFree(d_in_1);
  hipFree(d_in_2);
  hipFree(d_out);
}

//MOST LIKELY WORKS AS INTENDED AND HARD TO DEBUG
void sumParts(float *out, float *in, int len)
{
  // Pointer to device array
  float *d_in = 0;

  // Accumulating variable
  float *d_accum = 0;

  // Allocate memory for device arrays
  hipMalloc(&d_in, len*sizeof(float));
  hipMalloc(&d_accum, sizeof(float));

  // Copy from host to device
  hipMemcpy(d_in, in, len*sizeof(float), hipMemcpyHostToDevice);
  
  // Create d_accum
  hipMemset(d_accum, 0, sizeof(int));

  // Launch kernel to compute and store
  sumPartsKernel<<<len/TPB, TPB>>>(d_accum, d_in);
  
  // Copy from device to host
  hipMemcpy(out, d_accum, sizeof(float), hipMemcpyDeviceToHost);

  // Free the memory
  hipFree(d_in);
  hipFree(d_accum);
}
